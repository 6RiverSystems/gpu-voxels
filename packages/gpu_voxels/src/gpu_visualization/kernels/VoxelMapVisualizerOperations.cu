#include "hip/hip_runtime.h"
// this is for emacs file handling -*- mode: c++; indent-tabs-mode: nil -*-

// -- BEGIN LICENSE BLOCK ----------------------------------------------
// -- END LICENSE BLOCK ------------------------------------------------

//----------------------------------------------------------------------
/*!\file
 *
 * \author  Matthias Wagner
 * \date    2013-12-17
 *
 */
//----------------------------------------------------------------------/*
#include "VoxelMapVisualizerOperations.h"
#include <gpu_voxels/voxelmap/VoxelMap.hpp>

namespace gpu_voxels {
namespace visualization {

//////////////////////////////////// CUDA device functions /////////////////////////////////////////

//////////////////////////////////// CUDA kernel functions /////////////////////////////////////////

/**
 * Search the voxel map for occupied voxels and write the position into the VBO.
 * A voxel is occupied if its occupancy is greater than the occupancy_threshold.
 * start_voxel and end_voxel define a cuboid in the voxel map, which will be traversed (the rest of the map will be ignored).
 *
 * @param voxelMap: the device pointer of the voxel map
 * @param dim_voxel_map: the dimension of the voxel map
 * @param start_voxel: the position of the first voxel of the voxel map
 * @param end_voxel: the position of the last voxel of the voxel map
 * @param occupancy_threshold: the minimum occupancy value.
 * @param vbo: the device pointer of the VBO.
 * @param vbo_offsets: the offsets of the imaginary VBO segments.
 * @param vbo_limits: the maximum number of elements in the VBO segments.
 * @param write_index: the atomic counters for each voxel type (should be initialized with 0).
 * @param draw_voxel_type: if 0 the corresponding type at this index will not be drawn.
 * @param prefixes: stores the index of the VBO segment for each voxel type.
 */__global__ void fill_vbo_without_precounting(voxelmap::ProbabilisticVoxel* voxelMap, Vector3ui dim_voxel_map,
                                              Vector3ui dim_super_voxel, Vector3ui start_voxel,
                                              Vector3ui end_voxel, uint8_t occupancy_threshold, float4* vbo,
                                              uint32_t* vbo_offsets, uint32_t* vbo_limits,
                                              uint32_t* write_index, uint8_t* draw_voxel_type,
                                              uint8_t* prefixes)
{
  // Grid-Stride Loops
  for (uint32_t x = dim_super_voxel.x * (blockIdx.x * blockDim.x + threadIdx.x) + start_voxel.x;
      x < dim_voxel_map.x && x < end_voxel.x; x += blockDim.x * gridDim.x * dim_super_voxel.x)
  {
    for (uint32_t y = dim_super_voxel.y * (blockIdx.y * blockDim.y + threadIdx.y) + start_voxel.y;
        y < dim_voxel_map.y && y < end_voxel.y; y += blockDim.y * gridDim.y * dim_super_voxel.y)
    {
      for (uint32_t z = dim_super_voxel.z * (blockIdx.z * blockDim.z + threadIdx.z) + start_voxel.z;
          z < dim_voxel_map.z && z < end_voxel.z; z += blockDim.z * gridDim.z * dim_super_voxel.z)
      {
        bool found = false;

        // check if one of the voxel of the super voxel is occupied
        // these 3 loop are were slow for big super voxel sizes
        for (uint32_t i = x; i < dim_super_voxel.x + x && i < dim_voxel_map.x && !found; i++)
        {
          for (uint32_t j = y; j < dim_super_voxel.y + y && j < dim_voxel_map.y && !found; j++)
          {
            for (uint32_t k = z; k < dim_super_voxel.z + z && k < dim_voxel_map.z && !found; k++)
            {
              uint32_t index = 0xffff;
              uint8_t prefix;
              voxelmap::ProbabilisticVoxel voxel = voxelMap[k * dim_voxel_map.x * dim_voxel_map.y
                  + j * dim_voxel_map.x + i];

              if (voxel.getOccupancy() >= voxelmap::probability(occupancy_threshold - 128)) // Use signed values. Quick fix
              {
                //printf("occ thresh %u \n", occupancy_threshold);
                //map the occupancy on the first 10 types, so type element [0,9]
                uint8_t type = 9 - ((voxel.getOccupancy() + 128) * (10.f / 256.f));
                //printf("type %u \n", type);
                //type = 245 + type; // to match the other types use types from 245 - 254
                if (draw_voxel_type[type])
                {
                  prefix = prefixes[type];
                  index = atomicAdd(write_index + prefix, 1);
                  found = true;
                }
                if (index != 0xffff && index < vbo_limits[prefix])
                {
                  index = index + vbo_offsets[prefix];

                  vbo[index] = make_float4(x, z, y, dim_super_voxel.x);
                  // write the lower left front corner of the super voxel into the vbo as its translation
                  // use the z as height so switch z and y
                  found = true;
                }
              }
            }
          }
        }
      }
    }
  }
}

/**
 * Search the voxel map for occupied voxels and write the position into the VBO.
 * A voxel is occupied if its occupancy is greater than the occupancy_threshold.
 * start_voxel and end_voxel define a cuboid in the voxel map, which will be traversed (the rest of the map will be ignored).
 *
 * @param voxelMap: the device pointer of the voxel map
 * @param dim_voxel_map: the dimension of the voxel map
 * @param start_voxel: the position of the first voxel of the voxel map
 * @param end_voxel: the position of the last voxel of the voxel map
 * @param occupancy_threshold: the minimum occupancy value.
 * @param vbo: the device pointer of the VBO.
 * @param vbo_offsets: the offsets of the imaginary VBO segments.
 * @param vbo_limits: the maximum number of elements in the VBO segments.
 * @param write_index: the atomic counters for each voxel type (should be initialized with 0).
 * @param draw_voxel_type: if 0 the corresponding type at this index will not be drawn.
 * @param prefixes: stores the index of the VBO segment for each voxel type.
 */__global__ void fill_vbo_without_precounting(voxelmap::BitVectorVoxel* voxelMap, Vector3ui dim_voxel_map,
                                              Vector3ui dim_super_voxel, Vector3ui start_voxel,
                                              Vector3ui end_voxel, uint8_t occupancy_threshold, float4* vbo,
                                              uint32_t* vbo_offsets, uint32_t* vbo_limits,
                                              uint32_t* write_index, uint8_t* draw_voxel_type,
                                              uint8_t* prefixes)
{
  // Grid-Stride Loops
  for (uint32_t x = dim_super_voxel.x * (blockIdx.x * blockDim.x + threadIdx.x) + start_voxel.x;
      x < dim_voxel_map.x && x < end_voxel.x; x += blockDim.x * gridDim.x * dim_super_voxel.x)
  {
    for (uint32_t y = dim_super_voxel.y * (blockIdx.y * blockDim.y + threadIdx.y) + start_voxel.y;
        y < dim_voxel_map.y && y < end_voxel.y; y += blockDim.y * gridDim.y * dim_super_voxel.y)
    {
      for (uint32_t z = dim_super_voxel.z * (blockIdx.z * blockDim.z + threadIdx.z) + start_voxel.z;
          z < dim_voxel_map.z && z < end_voxel.z; z += blockDim.z * gridDim.z * dim_super_voxel.z)
      {
        bool found = false;

        // check if one of the voxel of the super voxel is occupied
        // these 3 loop are were slow for big super voxel sizes
        for (uint32_t i = x; i < dim_super_voxel.x + x && i < dim_voxel_map.x && !found; i++)
        {
          for (uint32_t j = y; j < dim_super_voxel.y + y && j < dim_voxel_map.y && !found; j++)
          {
            for (uint32_t k = z; k < dim_super_voxel.z + z && k < dim_voxel_map.z && !found; k++)
            {
              voxelmap::BitVectorVoxel voxel = voxelMap[k * dim_voxel_map.x * dim_voxel_map.y + j * dim_voxel_map.x + i];

              if (!voxel.bitVector().isZero())
              {
                for (uint32_t t = 0 ; t < min((unsigned long long) voxelmap::BIT_VECTOR_LENGTH, (unsigned long long) MAX_DRAW_TYPES); ++t)
                {
                  uint32_t index = 0xffff;
                  uint8_t prefix;

                  // show all swept volumes
                  if(t >= eVT_SWEPT_VOLUME_START && draw_voxel_type[eVT_SWEPT_VOLUME_START] && voxel.bitVector().getBit(t))
                  {
                    prefix = prefixes[eVT_SWEPT_VOLUME_START];
                    index = atomicAdd(write_index + prefix, 1);
                    found = true;
                  }
                  else if(draw_voxel_type[t] && voxel.bitVector().getBit(t))
                  {
                    prefix = prefixes[t];
                    index = atomicAdd(write_index + prefix, 1);
                    found = true;
                  }

//                 if (draw_voxel_type[t] && voxel.bitVector().getBit(t))
//                  {
//                    prefix = prefixes[t];
//                    index = atomicAdd(write_index + prefix, 1);
//                    found = true;
//                  }

                  if (index != 0xffff && index < vbo_limits[prefix])
                  {
                    index = index + vbo_offsets[prefix];

                    vbo[index] = make_float4(x, z, y, dim_super_voxel.x);
                    // write the lower left front corner of the super voxel into the vbo as its translation
                    // use the z as height so switch z and y
                    found = true;
                  }
                  if (found)
                  { // if a set bit in the bit vector was found leave this loop
                    break;
                  }
                }
              }

//              if ( && voxel->occupancy >= occupancy_threshold)
//              {
//                prefix = prefixes[voxel->voxeltype];
//                index = atomicAdd(write_index + prefix, 1);
//              }
//              if (index != 0xffff && index < vbo_limits[prefix])
//              {
//                index = index + vbo_offsets[prefix];
//
//                vbo[index] = make_float4(x, z, y, dim_super_voxel.x);
//                // write the lower left front corner of the super voxel into the vbo as its translation
//                // use the z as height so switch z and y
//                found = true;
//                break;
//
//              }
            }
          }
        }
      }
    }
  }
}

/**
 * Write the position of each cube into the VBO.
 *
 * @param cubes: the device pointer of the cube list.
 * @param size: the size of cubes.
 * @param vbo: the device pointer of the VBO.
 * @param vbo_offsets: the offsets of the imaginary VBO segments.
 * @param write_index: the atomic counters for each type (should be initialized with 0).
 * @param draw_voxel_type: if 0 the corresponding type at this index will not be drawn.
 * @param prefixes: stores the index of the VBO segment for each voxel type.
 */__global__ void fill_vbo_with_octree(Cube* cubes, uint32_t size, float4* vbo, uint32_t* vbo_offsets,
                                      uint32_t* write_index, uint8_t* draw_voxel_type, uint8_t* prefixes)
{
  //use Grid-Stride Loops
  for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x)
  {
    Cube* cube = cubes + i;
    uint32_t index = 0xffff;
    if (draw_voxel_type[cube->m_type])
    {
      uint8_t prefix = prefixes[cube->m_type];
      index = atomicAdd(write_index + prefix, 1);
      index = index + vbo_offsets[prefix];
      float x = cube->m_position.x;
      float y = cube->m_position.y;
      float z = cube->m_position.z;
      float w = cube->m_side_length;
      vbo[index] = make_float4(x, z, y, w);
      // write the position and the scale factor into the vbo
      // use the z as height so switch z and y
    }
  }
}

/**
 * Calculate the amount of cubes per type in the cubes list.
 *
 * @param cubes: the device pointer of the cube list.
 * @param size: the size of cubes.
 * @param cubes_per_type: Will contain the number of cubes per type afterwards (should be initialized with 0).
 * @param draw_voxel_type: if 0 the corresponding type at this index will not be drawn.
 * @param prefixes: stores the index of the VBO segment for each voxel type.
 */__global__ void calculate_cubes_per_type(Cube* cubes, uint32_t size, uint32_t* cubes_per_type,
                                          uint8_t* draw_voxel_type, uint8_t* prefixes)
{
  for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x)
  {
    Cube* cube = cubes + i;
    if (draw_voxel_type[cube->m_type])
    {
      uint8_t prefix = prefixes[cube->m_type];
      atomicAdd(cubes_per_type + prefix, 1);
    }
  }
}

} // end of ns
} // end of ns
