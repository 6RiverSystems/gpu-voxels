#include "hip/hip_runtime.h"
// this is for emacs file handling -*- mode: c++; indent-tabs-mode: nil -*-

// -- BEGIN LICENSE BLOCK ----------------------------------------------
// -- END LICENSE BLOCK ------------------------------------------------

//----------------------------------------------------------------------
/*!\file
 *
 * \author  Matthias Wagner
 * \date    2013-12-17
 *
 */
//----------------------------------------------------------------------/*
#include "VoxelMapVisualizerOperations.h"
#include <gpu_voxels/voxelmap/VoxelMap.hpp>

namespace gpu_voxels {
namespace visualization {

//////////////////////////////////// CUDA device functions /////////////////////////////////////////

//////////////////////////////////// CUDA kernel functions /////////////////////////////////////////

/**
 * Search the voxel map for occupied voxels and write the position into the VBO.
 * A voxel is occupied if its occupancy is greater than the occupancy_threshold.
 * start_voxel and end_voxel define a cuboid in the voxel map, which will be traversed (the rest of the map will be ignored).
 *
 * @param voxelMap: the device pointer of the voxel map
 * @param dim_voxel_map: the dimension of the voxel map
 * @param start_voxel: the position of the first voxel of the voxel map
 * @param end_voxel: the position of the last voxel of the voxel map
 * @param occupancy_threshold: the minimum occupancy value.
 * @param vbo: the device pointer of the VBO.
 * @param vbo_offsets: the offsets of the imaginary VBO segments.
 * @param vbo_limits: the maximum number of elements in the VBO segments.
 * @param write_index: the atomic counters for each voxel type (should be initialized with 0).
 * @param draw_voxel_type: if 0 the corresponding type at this index will not be drawn.
 * @param prefixes: stores the index of the VBO segment for each voxel type.
 */
__global__ void fill_vbo_without_precounting(ProbabilisticVoxel* voxelMap, Vector3ui dim_voxel_map,
                                              Vector3ui dim_super_voxel, Vector3ui start_voxel,
                                              Vector3ui end_voxel, Probability occupancy_threshold, float4* vbo,
                                              uint32_t* vbo_offsets, uint32_t* vbo_limits,
                                              uint32_t* write_index, uint8_t* draw_voxel_type,
                                              uint8_t* prefixes)
{
  // Grid-Stride Loops
  for (uint32_t x = dim_super_voxel.x * (blockIdx.x * blockDim.x + threadIdx.x) + start_voxel.x;
      x < dim_voxel_map.x && x < end_voxel.x; x += blockDim.x * gridDim.x * dim_super_voxel.x)
  {
    for (uint32_t y = dim_super_voxel.y * (blockIdx.y * blockDim.y + threadIdx.y) + start_voxel.y;
        y < dim_voxel_map.y && y < end_voxel.y; y += blockDim.y * gridDim.y * dim_super_voxel.y)
    {
      for (uint32_t z = dim_super_voxel.z * (blockIdx.z * blockDim.z + threadIdx.z) + start_voxel.z;
          z < dim_voxel_map.z && z < end_voxel.z; z += blockDim.z * gridDim.z * dim_super_voxel.z)
      {
        bool found = false;

        // check if one of the voxel of the super voxel is occupied
        // these 3 loop are were slow for big super voxel sizes
        for (uint32_t i = x; i < dim_super_voxel.x + x && i < dim_voxel_map.x && !found; i++)
        {
          for (uint32_t j = y; j < dim_super_voxel.y + y && j < dim_voxel_map.y && !found; j++)
          {
            for (uint32_t k = z; k < dim_super_voxel.z + z && k < dim_voxel_map.z && !found; k++)
            {
              uint32_t index = 0xffff;
              uint8_t prefix;
              ProbabilisticVoxel voxel = voxelMap[k * dim_voxel_map.x * dim_voxel_map.y
                  + j * dim_voxel_map.x + i];

              if (voxel.getOccupancy() >= occupancy_threshold)
              {
                //printf("occ thresh %u \n", occupancy_threshold);
                // map the occupancy on the SweptVolume types, which is a bit fuzzy, was only 250 Types are available
                // so we have to cap it
                uint8_t type = MIN((eBVM_SWEPT_VOLUME_START + voxel.getOccupancy()), eBVM_SWEPT_VOLUME_END);
                //printf("type %u \n", type);
                if (draw_voxel_type[type])
                {
                  prefix = prefixes[type];
                  index = atomicAdd(write_index + prefix, 1);
                  found = true;
                }
                if (index != 0xffff && index < vbo_limits[prefix])
                {
                  index = index + vbo_offsets[prefix];

                  vbo[index] = make_float4(x, y, z, dim_super_voxel.x);
                  // write the lower left front corner of the super voxel into the vbo as its translation
                  // use the z as height so switch z and y
                  found = true;
                }
              }
            }
          }
        }
      }
    }
  }
}

/**
 * Search the voxel map for occupied voxels and write the position into the VBO.
 * A voxel is occupied if its occupancy is greater than the occupancy_threshold.
 * start_voxel and end_voxel define a cuboid in the voxel map, which will be traversed (the rest of the map will be ignored).
 *
 * @param voxelMap: the device pointer of the voxel map
 * @param dim_voxel_map: the dimension of the voxel map
 * @param start_voxel: the position of the first voxel of the voxel map
 * @param end_voxel: the position of the last voxel of the voxel map
 * @param occupancy_threshold: the minimum occupancy value.
 * @param vbo: the device pointer of the VBO.
 * @param vbo_offsets: the offsets of the imaginary VBO segments.
 * @param vbo_limits: the maximum number of elements in the VBO segments.
 * @param write_index: the atomic counters for each voxel type (should be initialized with 0).
 * @param draw_voxel_type: if 0 the corresponding type at this index will not be drawn.
 * @param prefixes: stores the index of the VBO segment for each voxel type.
 */__global__ void fill_vbo_without_precounting(BitVectorVoxel* voxelMap, Vector3ui dim_voxel_map,
                                              Vector3ui dim_super_voxel, Vector3ui start_voxel,
                                              Vector3ui end_voxel, uint8_t occupancy_threshold, float4* vbo,
                                              uint32_t* vbo_offsets, uint32_t* vbo_limits,
                                              uint32_t* write_index, uint8_t* draw_voxel_type,
                                              uint8_t* prefixes)
{
  // Grid-Stride Loops
  for (uint32_t x = dim_super_voxel.x * (blockIdx.x * blockDim.x + threadIdx.x) + start_voxel.x;
      x < dim_voxel_map.x && x < end_voxel.x; x += blockDim.x * gridDim.x * dim_super_voxel.x)
  {
    for (uint32_t y = dim_super_voxel.y * (blockIdx.y * blockDim.y + threadIdx.y) + start_voxel.y;
        y < dim_voxel_map.y && y < end_voxel.y; y += blockDim.y * gridDim.y * dim_super_voxel.y)
    {
      for (uint32_t z = dim_super_voxel.z * (blockIdx.z * blockDim.z + threadIdx.z) + start_voxel.z;
          z < dim_voxel_map.z && z < end_voxel.z; z += blockDim.z * gridDim.z * dim_super_voxel.z)
      {
        bool found = false;

        // check if one of the voxel of the super voxel is occupied
        // these 3 loop are were slow for big super voxel sizes
        for (uint32_t i = x; i < dim_super_voxel.x + x && i < dim_voxel_map.x && !found; i++)
        {
          for (uint32_t j = y; j < dim_super_voxel.y + y && j < dim_voxel_map.y && !found; j++)
          {
            for (uint32_t k = z; k < dim_super_voxel.z + z && k < dim_voxel_map.z && !found; k++)
            {
              BitVectorVoxel voxel = voxelMap[k * dim_voxel_map.x * dim_voxel_map.y + j * dim_voxel_map.x + i];

              if (!voxel.bitVector().isZero())
              {
                for (uint32_t t = 0 ; t < min((unsigned long long) BIT_VECTOR_LENGTH, (unsigned long long) MAX_DRAW_TYPES); ++t)
                {
                  uint32_t index = 0xffff;
                  uint8_t prefix;

                 if (draw_voxel_type[t] && voxel.bitVector().getBit(t))
                  {
                    prefix = prefixes[t];
                    index = atomicAdd(write_index + prefix, 1);
                    found = true;
                  }

                  if (index != 0xffff && index < vbo_limits[prefix])
                  {
                    index = index + vbo_offsets[prefix];

                    vbo[index] = make_float4(x, y, z, dim_super_voxel.x);
                    // write the lower left front corner of the super voxel into the vbo as its translation
                    // use the z as height so switch z and y
                    found = true;
                  }
                  if (found)
                  { // if a set bit in the bit vector was found leave this loop
                    break;
                  }
                }
              }

//              if ( && voxel->occupancy >= occupancy_threshold)
//              {
//                prefix = prefixes[voxel->voxeltype];
//                index = atomicAdd(write_index + prefix, 1);
//              }
//              if (index != 0xffff && index < vbo_limits[prefix])
//              {
//                index = index + vbo_offsets[prefix];
//
//                vbo[index] = make_float4(x, y, z, dim_super_voxel.x);
//                // write the lower left front corner of the super voxel into the vbo as its translation
//                // use the z as height so switch z and y
//                found = true;
//                break;
//
//              }
            }
          }
        }
      }
    }
  }
}

/**
 * Write the position of each cube into the VBO.
 *
 * @param cubes: the device pointer of the cube list.
 * @param size: the size of cubes.
 * @param vbo: the device pointer of the VBO.
 * @param vbo_offsets: the offsets of the imaginary VBO segments.
 * @param write_index: the atomic counters for each type (should be initialized with 0).
 * @param draw_voxel_type: if 0 the corresponding type at this index will not be drawn.
 * @param prefixes: stores the index of the VBO segment for each voxel type.
 */
__global__ void fill_vbo_with_cubelist(Cube* cubes, uint32_t size, float4* vbo, uint32_t* vbo_offsets,
                                      uint32_t* write_index, uint8_t* draw_voxel_type, uint8_t* prefixes)
{
  //use Grid-Stride Loops
  for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x)
  {
    Cube* cube = cubes + i;

    uint32_t index = 0xffff;

    bool found = false;
    for (size_t t = 0 ; t < MAX_DRAW_TYPES; ++t)
    {
      // TODO: Create a bitmask outside the kernel and just do a bit comparison in here! Instead of for-loop
      if (draw_voxel_type[t] && cube->m_type_vector.getBit(t))
      {
        uint8_t prefix = prefixes[t];
        index = atomicAdd(write_index + prefix, 1);
        index = index + vbo_offsets[prefix];
        // write the position and the scale factor into the vbo
        float x = cube->m_position.x;
        float y = cube->m_position.y;
        float z = cube->m_position.z;
        float w = cube->m_side_length;
        //            printf("Found voxel at (%f,%f,%f) with voxel type %lu, and sidelenth %f\n", x, y, z, t, w);
        vbo[index] = make_float4(x, y, z, w);
        found = true;
      }

      if (found)
      { // if a set bit in the bit vector was found leave this loop
        break;
      }
    }
  }
}

/**
 * Calculate the amount of cubes per type in the cubes list.
 *
 * @param cubes: the device pointer of the cube list.
 * @param size: the size of cubes.
 * @param cubes_per_type: Will contain the number of cubes per type afterwards (should be initialized with 0).
 * @param draw_voxel_type: if 0 the corresponding type at this index will not be drawn.
 * @param prefixes: stores the index of the VBO segment for each voxel type.
 */
__global__ void calculate_cubes_per_type_list(Cube* cubes, uint32_t size, uint32_t* cubes_per_type, uint8_t* draw_voxel_type, uint8_t* prefixes)
{
  for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x)
  {
    Cube* cube = cubes + i;

    bool found = false;
    // TODO: Create a bitmask outside the kernel and just do a bit comparison in here! Instead of for-loop
    for (size_t t = 0; t < MAX_DRAW_TYPES; ++t)
    {
      uint8_t prefix;
      if (draw_voxel_type[t] && cube->m_type_vector.getBit(t))
      {
        prefix = prefixes[t];
        atomicAdd(cubes_per_type + prefix, 1);
        //printf("Found voxel with type %lu. Now drawing %u voxels of type %lu\n", t, *(cubes_per_type+prefix), t);
        found = true;
      }

      if (found)
      { // if a set bit in the bit vector was found leave this loop
        break;
      }
    }
  }
}

} // end of ns
} // end of ns
