#include "hip/hip_runtime.h"
// this is for emacs file handling -*- mode: c++; indent-tabs-mode: nil -*-

// -- BEGIN LICENSE BLOCK ----------------------------------------------
// -- END LICENSE BLOCK ------------------------------------------------

//----------------------------------------------------------------------
/*!\file
 *
 * \author  Sebastian Klemm
 * \date    2012-09-13
 *
 */
//----------------------------------------------------------------------
//#define LOCAL_DEBUG
#undef LOCAL_DEBUG

#include "KinematicOperations.h"
#include <stdio.h>

namespace gpu_voxels {
__global__
void kernelUpdateTransformations(uint8_t chain_size, uint8_t* joint_types,
                                 KinematicLink::DHParameters* dh_parameters, Matrix4f* dh_transformations, Matrix4f* local_transformations)
{
  const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < chain_size)
  {
    Matrix4f matrix;
    Matrix4f matrix_local;

    convertDHtoM(dh_parameters[i].theta,
                 dh_parameters[i].d,
                 0, // currently only b = 0
                 dh_parameters[i].a,
                 dh_parameters[i].alpha,
                 dh_parameters[i].value,
                 joint_types[i],
                 matrix);

    dh_transformations[i] = matrix;

    /* local transformations use only d and theta part of DH-parameters.
     * and must be used for point cloud of the current joint
     */
    convertDHtoM(dh_parameters[i].theta,
                 dh_parameters[i].d,
                 0, // currently only b = 0
                 dh_parameters[i].a, 			// a = 0
                 dh_parameters[i].alpha, 		// alpha = 0
                 dh_parameters[i].value,
                 joint_types[i],
                 matrix_local);

    
    local_transformations[i] = matrix_local;


//    printf("dhtom: updated matrix %u \na11 .. a14:\t %f %f %f %f \na21 .. a24:\t %f %f %f %f \na31 .. a34:\t %f %f %f %f \na41 .. a44:\t %f %f %f %f\n",
//           i,
//           matrix.a11, matrix.a12, matrix.a13, matrix.a14,
//           matrix.a21, matrix.a22, matrix.a23, matrix.a24,
//           matrix.a31, matrix.a32, matrix.a33, matrix.a34,
//           matrix.a41, matrix.a42, matrix.a43, matrix.a44);
  }
}




__global__
void kernelKinematicChainTransform(uint8_t chain_size, uint8_t joint_to_transform, const Matrix4f* basis_transformation,
                                   const Matrix4f* dh_transformations, const Matrix4f* local_transformations,
                                   const uint32_t* point_cloud_sizes, const Vector3f** point_clouds, Vector3f** transformed_point_clouds)
{
	//this function is too slow, since every thread computes the transformation matrix
	//use the overloaded version with fewer parameters instead
  uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

  
   Matrix4f transformation;


//  if (i==1) printf("transform (%u): (%f, %f, %f, %f)\n", sizeof(transformation), transformation.a11, transformation.a12, transformation.a13, transformation.a14);

  if (joint_to_transform > 0)   // not basis
  {
	// set d and theta part for current joint
	transformation = local_transformations[joint_to_transform-1];

	// append full dh parameters of other joints
	for (uint32_t j = joint_to_transform-1; j>=1; j--)
	{
	   transformation = dh_transformations[j-1] * transformation;
//      transformation.leftMultiply(dh_transformations[j-1]);
	}
  }
  else  // basis
  {
	transformation.setIdentity();
  }	
//  transformation.leftMultiply(*basis_transformation);
  transformation = (*basis_transformation) * transformation;

  __syncthreads();

  // if more than max_nr_of_blocks points are in ptcloud we need a loop
  while (i < point_cloud_sizes[joint_to_transform])
  {
//    applyTransform(transformation, point_clouds[joint_to_transform][i], transformed_point_clouds[joint_to_transform][i]);
    transformed_point_clouds[joint_to_transform][i] = transformation * point_clouds[joint_to_transform][i];
//    if (i==1)
//      printf("transforming (%f, %f, %f) --> (%f, %f, %f)",
//             point_clouds[joint_to_transform][i].x, point_clouds[joint_to_transform][i].y, point_clouds[joint_to_transform][i].z,
//             transformed_point_clouds[joint_to_transform][i].x, transformed_point_clouds[joint_to_transform][i].y, transformed_point_clouds[joint_to_transform][i].z);
//     transformed_point_clouds[joint_to_transform][i] = transformation * point_clouds[joint_to_transform][i];

    // increment by number of all threads that are running
    i += blockDim.x * gridDim.x;
  }

}




__global__
void kernelKinematicChainTransform(uint8_t chain_size, uint8_t joint_to_transform, const Matrix4f* transformation,                                   
                                   const MetaPointCloudStruct *point_clouds, MetaPointCloudStruct *transformed_point_clouds)
{
  uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

// copying the transformation matrix to shared memory might be faster than accessing it from the global memory
// but the copying in the next lines doesnt work for some reason
//  __shared__ Matrix4f cache_transformation;
//    
//  if(threadIdx.x < 16)				//Matrix4f should contain 16 * 4 bytes of memory
//  {
//	  uint32_t* ptr1 = (uint32_t*)(&cache_transformation);
//	  uint32_t* ptr2 = (uint32_t*)transformation;
////	  ptr1 += threadIdx.x;
////	  ptr2 += threadIdx.x;
//	  
//	  
//	  *(ptr1 + threadIdx.x) = *(ptr2 + threadIdx.x);
//  }
//  __syncthreads();  
  
//  if (i==1) printf("transform (%u): (%f, %f, %f, %f)\n", sizeof(transformation), transformation.a11, transformation.a12, transformation.a13, transformation.a14);

  // if more than max_nr_of_blocks points are in ptcloud we need a loop
  while (i < point_clouds->cloud_sizes[joint_to_transform])
  {

//    applyTransform(transformation, point_clouds[joint_to_transform][i], transformed_point_clouds[joint_to_transform][i]);
    transformed_point_clouds->clouds_base_addresses[joint_to_transform][i] = (*transformation) * point_clouds->clouds_base_addresses[joint_to_transform][i];
//    if (i==1)
//      printf("transforming (%f, %f, %f) --> (%f, %f, %f)",
//             point_clouds[joint_to_transform][i].x, point_clouds[joint_to_transform][i].y, point_clouds[joint_to_transform][i].z,
//             transformed_point_clouds[joint_to_transform][i].x, transformed_point_clouds[joint_to_transform][i].y, transformed_point_clouds[joint_to_transform][i].z);
//     transformed_point_clouds[joint_to_transform][i] = transformation * point_clouds[joint_to_transform][i];

    // increment by number of all threads that are running
    i += blockDim.x * gridDim.x;
  }
}



__global__
void kernelTransformPoseAlongChain(uint8_t chain_size, uint8_t joint_to_transform,
                                   const Matrix4f* basis_transformation, Matrix4f* dh_transformations,
                                   Vector3f* dev_point, Vector3f* dev_result)
{
  uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i==0)
  {
    Matrix4f transformation;
    transformation.setIdentity();

    for (uint32_t j=1; j<=joint_to_transform; j++)
    {
      transformation = transformation * dh_transformations[j-1];
      printf("executing for loop : %u\n", j);
    }
    transformation = (*basis_transformation) * transformation;

    (*dev_result) = transformation * (*dev_point);
  }
  else
  {
    printf("kernelTransformPoseAlongChain: This kernel should be configured to run with 1 thread only!\n");
  }

}




} // end of namespace gpu_voxels
#ifdef LOCAL_DEBUG
#undef LOCAL_DEBUG
#endif
