// this is for emacs file handling -*- mode: c++; indent-tabs-mode: nil -*-

// -- BEGIN LICENSE BLOCK ----------------------------------------------
// -- END LICENSE BLOCK ------------------------------------------------

//----------------------------------------------------------------------
/*!\file
 *
 * \author  Sebastian Klemm
 * \date    2012-08-31
 *
 *
 */
//----------------------------------------------------------------------
#include "cuda_handling.h"

namespace gpu_voxels {

bool cuCheckForError(const char* file, int line)
{
  hipError_t cuda_error = hipGetLastError();
  if (cuda_error != hipSuccess)
  {
    LOGGING_ERROR(CudaLog,
                  hipGetErrorString(cuda_error) << "(" << cuda_error << ") in " << file << " on line " << line << "." << endl);
    return false;
  }
  return true;
}


bool cuHandleError(hipError_t cuda_error, const char* file, int line)
{
  if (cuda_error != hipSuccess)
  {
    LOGGING_ERROR(CudaLog,
                  hipGetErrorString(cuda_error) << " in " << file << " on line " << line << "." << endl);
    return false;
  }
  return true;
}

bool cuGetNrOfDevices(int* nr_of_devices)
{
  if (!HANDLE_CUDA_ERROR(hipGetDeviceCount(nr_of_devices)))
  {
    return false;
  }
  return true;
}

bool cuGetDeviceInfo(hipDeviceProp_t* device_properties, int nr_of_devices)
{
  for (int i = 0; i < nr_of_devices; i++)
  {
    if (!HANDLE_CUDA_ERROR(hipGetDeviceProperties(&device_properties[i] , i)))
    {
      return false;
    }
  }
  return true;
}

bool cuTestAndInitDevice()
{

  // The test requires an architecture SM25 or greater (CDP capable).

  int device_count = 0, device = -1;
  cuGetNrOfDevices(&device_count);
  for (int i = 0; i < device_count; ++i)
  {
    hipDeviceProp_t properties;
    HANDLE_CUDA_ERROR(hipGetDeviceProperties(&properties, i));
    if (properties.major > 2 || (properties.major == 2 && properties.minor >= 0))
    {
      device = i;
      LOGGING_INFO(CudaLog, "Running on GPU " << i << " (" << properties.name << ")" << endl);
      break;
    }
  }
  if (device == -1)
  {
    std::cerr << "No device with SM 2.5 or higher found, which is required for GPU-Voxels.\n"
        << std::endl;
    return false;
  }
  hipSetDevice(device);
  HANDLE_CUDA_ERROR(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
//HANDLE_CUDA_ERROR(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
  return true;
}

void cuPrintDeviceMemoryInfo()
{
  HANDLE_CUDA_ERROR(hipDeviceSynchronize());
  //unsigned int free, total, used;
  size_t free, total, used;
  hipMemGetInfo(&free, &total);
  used = total - free;

  const float byte2mb = (float) 1 / (1024.0 * 1024.0);

  LOGGING_INFO(CudaLog, "Device memory status:" << endl);
  LOGGING_INFO(CudaLog, "-----------------------------------" << endl);
  LOGGING_INFO(CudaLog, "total memory (MB)  : " << (float) total * byte2mb << endl);
  LOGGING_INFO(CudaLog, "free  memory (MB)  : " << (float) free * byte2mb << endl);
  LOGGING_INFO(CudaLog, "used  memory (MB)  : " << (float) used * byte2mb << endl);
  LOGGING_INFO(CudaLog, "-----------------------------------" << endl);
}

} // end of namespace
